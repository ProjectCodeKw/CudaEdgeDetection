
#include "hip/hip_runtime.h"

#include <cstdlib> // For system()
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <time.h>
#include <math.h>
#include <cstdlib> // For exit()
using namespace std;

#define BLOCK_SIZE 16 // 2D block size

// Function to read PPM file (Grayscale format P5)
vector<unsigned char> readPPM(const string &filename, int &width, int &height)
{
    ifstream file(filename, ios::binary);
    if (!file)
    {
        cerr << "Error opening PPM file\n";
        exit(1);
    }

    string magic;
    file >> magic; // Read P5
    if (magic != "P5")
    {
        cerr << "Invalid PPM format\n";
        exit(1);
    }

    // Read width, height, and max color value
    file >> width >> height;
    int maxVal;
    file >> maxVal;
    file.ignore(1); // Consume newline before pixel data

    // Read pixel data (grayscale)
    vector<unsigned char> pixelData(width * height);
    file.read(reinterpret_cast<char *>(pixelData.data()), pixelData.size());

    return pixelData;
}

void CPU_EDGEDETECTION(unsigned char *a, unsigned char *c, int width, int height)
{
    for (int y = 0; y < height - 1; y++)
    {
        for (int x = 0; x < width - 1; x++)
        {
            float f_xy1 = a[y * width + (x + 1)];        // f(x, y+1)
            float f_x1y = a[(y + 1) * width + x];        // f(x+1, y)
            float f_x1y1 = a[(y + 1) * width + (x + 1)]; // f(x+1, y+1)
            float f_xy = a[y * width + x];               // f(x, y)
            float gx = f_xy1 - f_x1y;                    // (f(x,y+1) - f(x+1,y))
            float gy = f_x1y1 - f_xy;                    // (f(x+1,y+1) - f(x,y))
            c[y * width + x] = sqrtf(gx * gx + gy * gy);
        }
    }
}

__global__ void EDGEDETECTION(unsigned char *a, unsigned char *c, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x; // thread location
    int y = blockIdx.y * blockDim.y + threadIdx.y; // thread location

    if (x < width - 1 && y < height - 1)

    {
        float f_xy1 = a[y * width + (x + 1)];        // f(x, y+1)
        float f_x1y = a[(y + 1) * width + x];        // f(x+1, y)
        float f_x1y1 = a[(y + 1) * width + (x + 1)]; // f(x+1, y+1)
        float f_xy = a[y * width + x];               // f(x, y)

        float gx = f_xy1 - f_x1y; // (f(x,y+1) - f(x+1,y))
        float gy = f_x1y1 - f_xy; // (f(x+1,y+1) - f(x,y))

        c[y * width + x] = sqrtf(gx * gx + gy * gy);
    }
}

int main()
{

    // read the ppm file
    string path = "og_image.ppm";
    int width, height;
    vector<unsigned char> h_a = readPPM(path, width, height);
    size_t size = h_a.size();

    // allocatin memory for the sizes in the GPU
    unsigned char *d_a, *d_c;
    hipMalloc(&d_a, size);           // input image
    hipMalloc(&d_c, width * height); // output edge

    // copy to the GPU
    hipMemcpy(d_a, h_a.data(), size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);                                       // Block dimensions in 2D
    dim3 gridSize((int)ceil(width / BLOCK_SIZE), (int)ceil(height / BLOCK_SIZE)); // width, height

    clock_t start_cpu = clock();
    CPU_EDGEDETECTION(h_a.data(), h_a.data(), width, height);
    clock_t end = clock();
    double cpu_time_used = ((double)(end - start_cpu)) / CLOCKS_PER_SEC;
    printf("CPU took: %f ms to execute, \n", cpu_time_used * 1000);

    hipEvent_t start, stop; // for GPU timing
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    EDGEDETECTION<<<gridSize, blockSize>>>(d_a, d_c, width, height);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    float milliseconds1 = 0;
    hipEventElapsedTime(&milliseconds1, start, stop);
    printf("GPU time: %0.6f ms\n", milliseconds1);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }

    // Copy result back to host
    vector<unsigned char> h_c(width * height);
    hipMemcpy(h_c.data(), d_c, width * height, hipMemcpyDeviceToHost);

    // Save output to file (Binary PPM P5) and overwrite it if it exists
    ofstream outFile("edge_result.ppm", ios::binary | ios::trunc);
    outFile << "P5\n"
            << width << " " << height << "\n255\n";
    outFile.write(reinterpret_cast<char *>(h_c.data()), h_c.size());
    outFile.close();

    printf("SPEED UP: %0.2f\n", (cpu_time_used * 1000) / milliseconds1);

    // Free memory
    hipFree(d_a);
    hipFree(d_c);

    exit(0);
}
